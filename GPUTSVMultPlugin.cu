#include <emmintrin.h>
#include <sys/time.h> 
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <iostream>
#include <iomanip>
#include <fstream>

#include "GPUTSVMultPlugin.h"

void GPUTSVMultPlugin::input(std::string file) {
 inputfile = file;
 std::ifstream ifile(inputfile.c_str(), std::ios::in);
 while (!ifile.eof()) {
   std::string key, value;
   ifile >> key;
   ifile >> value;
   parameters[key] = value;
 }
 M = atoi(parameters["M"].c_str());
 N = atoi(parameters["N"].c_str());
 P = atoi(parameters["P"].c_str());
 A = (double*) malloc(N*N*sizeof(double));
 B = (double*) malloc(N*N*sizeof(double));
 C = (double*) malloc(N*N*sizeof(double));
 std::ifstream myinput((std::string(PluginManager::prefix())+parameters["matrix1"]).c_str(), std::ios::in);
 int i;
 for (i = 0; i < M*N; ++i) {
	int k;
	myinput >> k;
        A[i] = k;
 }
 std::ifstream myinput2((std::string(PluginManager::prefix())+parameters["matrix2"]).c_str(), std::ios::in);
 for (i = 0; i < N*P; ++i) {
	int k;
	myinput2 >> k;
        B[i] = k;
 }
}




void GPUTSVMultPlugin::run() {
	double *pA;
	double *pB;
	double *pC;
hipMalloc((void**)&pA, (M*N)*sizeof(double));
hipMalloc((void**)&pB, (N*P)*sizeof(double));
hipMalloc((void**)&pC, (M*P)*sizeof(double));
hipMemcpy(pA, A, (M*N)*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(pB, B, (N*P)*sizeof(double), hipMemcpyHostToDevice);
printf("***Mult on %d x %d Matrix on GPU***\n",N,N);
MatMult<<<M,P>>>(pA, pB, pC, M, N, P);
hipMemcpy(C, pC, (M*P)*sizeof(double), hipMemcpyDeviceToHost);

hipFree(pA);
hipFree(pB);
hipFree(pC);

}

void GPUTSVMultPlugin::output(std::string file) {
	std::ofstream outfile(file.c_str(), std::ios::out);
        int i, j;
        for (i = 0; i < M; ++i){
            for (j = 0; j < P; ++j){
		outfile << C[i*P+j];//std::setprecision(0) << a[i*N+j];
		if (j != P-1)
			outfile << "\t";
		else
			outfile << "\n";
            }
	}
	free(A);
	free(B);
	free(C);
}



PluginProxy<GPUTSVMultPlugin> GPUTSVMultPluginProxy = PluginProxy<GPUTSVMultPlugin>("GPUTSVMult", PluginManager::getInstance());


